#include "hip/hip_runtime.h"
#include "mex.h"
#include <hip/hip_runtime.h>
#include <math.h>

// RK4의 각 단계에서 미분값(dx/dt, dy/dt)을 계산하는 디바이스 함수
__device__ void calculate_derivatives(
    double* dx_out, double* dy_out,
    const double* x_in, const double* y_in,
    const double* alpha_x, const double* beta_x,
    const double* alpha_y, const double* beta_y,
    double ext_force, int N, int tid)
{
    double dx = 0.0;
    double dy = 0.0;
    for (int j = 0; j < N; ++j) {
        dx += alpha_x[j * N + tid] * x_in[j];
        dx += beta_x[j * N + tid]  * y_in[j] * (1.0 - fabs(y_in[j]));
        dy += alpha_y[j * N + tid] * y_in[j];
        dy += beta_y[j * N + tid]  * x_in[j] * (1.0 - fabs(x_in[j]));
    }
    dx += ext_force;
    dy += ext_force;
    dx_out[tid] = dx;
    dy_out[tid] = dy;
}


__global__ void model2_kernel(
    double* d_X, double* d_Y,
    double* d_x, double* d_y,
    const double* alpha_x, const double* beta_x,
    const double* alpha_y, const double* beta_y,
    double ext_force_amp, double ext_force_freq, double phase_shift,
    double dt, int numSteps, int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= N) return;

    // RK4 중간 계산을 위한 공유 메모리 할당
    extern __shared__ double temp_storage[];
    double* k1_x = &temp_storage[0 * N];
    double* k1_y = &temp_storage[1 * N];
    double* k2_x = &temp_storage[2 * N];
    double* k2_y = &temp_storage[3 * N];
    double* k3_x = &temp_storage[4 * N];
    double* k3_y = &temp_storage[5 * N];
    double* k4_x = &temp_storage[6 * N];
    double* k4_y = &temp_storage[7 * N];
    double* temp_x = &temp_storage[8 * N];
    double* temp_y = &temp_storage[9 * N];

    // 초기 상태 저장
    d_X[tid * numSteps] = d_x[tid];
    d_Y[tid * numSteps] = d_y[tid];
    __syncthreads();

    for (int step = 1; step < numSteps; ++step) {
        double t = (step - 1) * dt;

        // --- 4차 룽게-쿠타(RK4) 계산 시작 ---

        // k1 계산
        double ext_force_t0 = ext_force_amp * sin(ext_force_freq * t + phase_shift);
        calculate_derivatives(k1_x, k1_y, d_x, d_y, alpha_x, beta_x, alpha_y, beta_y, ext_force_t0, N, tid);
        __syncthreads();

        // k2 계산
        double ext_force_t1 = ext_force_amp * sin(ext_force_freq * (t + 0.5 * dt) + phase_shift);
        for (int j = 0; j < N; ++j) { temp_x[j] = d_x[j] + 0.5 * dt * k1_x[j]; temp_y[j] = d_y[j] + 0.5 * dt * k1_y[j]; }
        calculate_derivatives(k2_x, k2_y, temp_x, temp_y, alpha_x, beta_x, alpha_y, beta_y, ext_force_t1, N, tid);
        __syncthreads();

        // k3 계산
        for (int j = 0; j < N; ++j) { temp_x[j] = d_x[j] + 0.5 * dt * k2_x[j]; temp_y[j] = d_y[j] + 0.5 * dt * k2_y[j]; }
        calculate_derivatives(k3_x, k3_y, temp_x, temp_y, alpha_x, beta_x, alpha_y, beta_y, ext_force_t1, N, tid);
        __syncthreads();

        // k4 계산
        double ext_force_t2 = ext_force_amp * sin(ext_force_freq * (t + dt) + phase_shift);
        for (int j = 0; j < N; ++j) { temp_x[j] = d_x[j] + dt * k3_x[j]; temp_y[j] = d_y[j] + dt * k3_y[j]; }
        calculate_derivatives(k4_x, k4_y, temp_x, temp_y, alpha_x, beta_x, alpha_y, beta_y, ext_force_t2, N, tid);
        __syncthreads();

        // 최종 상태 업데이트
        d_x[tid] += (dt / 6.0) * (k1_x[tid] + 2.0 * k2_x[tid] + 2.0 * k3_x[tid] + k4_x[tid]);
        d_y[tid] += (dt / 6.0) * (k1_y[tid] + 2.0 * k2_y[tid] + 2.0 * k3_y[tid] + k4_y[tid]);
        __syncthreads();
        
        // 결과 저장
        d_X[tid * numSteps + step] = d_x[tid];
        d_Y[tid * numSteps + step] = d_y[tid];
    }
}


void mexFunction(int nlhs, mxArray* plhs[],
                 int nrhs, const mxArray* prhs[])
{
    if (nrhs != 12) {
        mexErrMsgIdAndTxt("simulateModel2_full:invalidNumInputs",
            "Expected 12 inputs: dt, numSteps, x0, y0, alpha_x, beta_x, alpha_y, beta_y, ext_force_amp, ext_force_freq, N, phase_shift");
    }

    double dt             = mxGetScalar(prhs[0]);
    int    numSteps       = (int)mxGetScalar(prhs[1]);
    double* h_x0          = mxGetPr(prhs[2]);
    double* h_y0          = mxGetPr(prhs[3]);
    double* h_alpha_x     = mxGetPr(prhs[4]);
    double* h_beta_x      = mxGetPr(prhs[5]);
    double* h_alpha_y     = mxGetPr(prhs[6]);
    double* h_beta_y      = mxGetPr(prhs[7]);
    double ext_force_amp  = mxGetScalar(prhs[8]);
    double ext_force_freq = mxGetScalar(prhs[9]);
    int    N              = (int)mxGetScalar(prhs[10]);
    double phase_shift    = mxGetScalar(prhs[11]); 

    plhs[0] = mxCreateDoubleMatrix(N, numSteps, mxREAL);
    plhs[1] = mxCreateDoubleMatrix(N, numSteps, mxREAL);
    double* h_X = mxGetPr(plhs[0]);
    double* h_Y = mxGetPr(plhs[1]);

    double *d_X, *d_Y, *d_x, *d_y, *d_alpha_x, *d_beta_x, *d_alpha_y, *d_beta_y;
    size_t stateSize  = N * numSteps * sizeof(double);
    size_t nodeSize   = N * sizeof(double);
    size_t matrixSize = N * N * sizeof(double);

    hipMalloc(&d_X, stateSize);
    hipMalloc(&d_Y, stateSize);
    hipMalloc(&d_x, nodeSize);
    hipMalloc(&d_y, nodeSize);
    hipMalloc(&d_alpha_x, matrixSize);
    hipMalloc(&d_beta_x,  matrixSize);
    hipMalloc(&d_alpha_y, matrixSize);
    hipMalloc(&d_beta_y,  matrixSize);

    hipMemcpy(d_x, h_x0, nodeSize, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y0, nodeSize, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha_x, h_alpha_x, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_beta_x,  h_beta_x,  matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha_y, h_alpha_y, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_beta_y,  h_beta_y,  matrixSize, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    
    // --- 수정된 부분: 커널 호출 시 공유 메모리 크기 지정 ---
    // RK4 계산에 필요한 10개의 임시 배열(크기 N)을 공유 메모리에 할당
    size_t shared_mem_size = 10 * N * sizeof(double);
    model2_kernel<<<numBlocks, blockSize, shared_mem_size>>>(d_X, d_Y, d_x, d_y,
                                                            d_alpha_x, d_beta_x,
                                                            d_alpha_y, d_beta_y,
                                                            ext_force_amp, ext_force_freq, phase_shift,
                                                            dt, numSteps, N);
    // ----------------------------------------------------

    hipMemcpy(h_X, d_X, stateSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_Y, d_Y, stateSize, hipMemcpyDeviceToHost);

    hipFree(d_X); hipFree(d_Y);
    hipFree(d_x); hipFree(d_y);
    hipFree(d_alpha_x); hipFree(d_beta_x);
    hipFree(d_alpha_y); hipFree(d_beta_y);
}